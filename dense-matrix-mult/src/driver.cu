#include "driver.h"
#include "kernels/kernels.h"
#include "util.hpp"
#include <chrono>
#include <hipblas.h>

float compute(std::vector<float> &C, const std::vector<float> &A, const std::vector<float> &B,
              const Configuration &config) {

    hipDeviceReset();
    CHECK_ERR;

    float *devA{nullptr}, *devB{nullptr}, *devC{nullptr};
    {
      // Allocate matrices A, B, C on device
      hipMalloc(&devA, A.size() * sizeof(float));
      hipMalloc(&devB, B.size() * sizeof(float));
      hipMalloc(&devC, C.size() * sizeof(float));
    }

    {
      // Copy the data from host to the device
      // NOTE: You may copy C as well, as it is zeroed, or hipMemset it to zero on the device
      hipMemcpy(devA, A.data(), A.size() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(devB, B.data(), B.size() * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(devC, C.data(), C.size() * sizeof(float), hipMemcpyHostToDevice);

    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float cpuTime{};
    hipEvent_t startTimer{}, stopTimer{};
    hipEventCreate(&startTimer);
    hipEventCreate(&stopTimer);


    // Start computing
    hipEventRecord(startTimer, 0);
    switch (config.kernelType) {
        case KernelType::KERNEL_CPU: {
            auto begin = std::chrono::high_resolution_clock::now();
            // NOTE: repeat loop is inside of cpu::matrixMult
            cpu::matrixMult(C, A, B, config);
            auto end = std::chrono::high_resolution_clock::now();
            cpuTime =
                std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(end - begin).count();
            break;
        }
        case KernelType::KERNEL_CUBLAS: {
            float alpha = 1.0f, beta = 1.0f;
            for (int i = 0; i < config.numRepeats; ++i) {
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, config.matrixSize, config.matrixSize,
                            config.matrixSize, &alpha, devA, config.matrixSize, devB,
                            config.matrixSize, &beta, devC, config.matrixSize);
            }
            CHECK_ERR;
            break;
        }
        default: {
            // NOTE: repeat loop is inside of gpu::matrixMult
            gpu::matrixMult(devA, devB, devC, config);
            break;
        }
    }

    hipEventRecord(stopTimer, 0);
    hipEventSynchronize(stopTimer);
    CHECK_ERR;
    // NOTE: hipEventSynchronize(stopTimer) is implicit hipDeviceSynchronize() in this context

    float gpuTime{};
    hipEventElapsedTime(&gpuTime, startTimer, stopTimer);

    // release resources
    hipblasDestroy(handle);
    hipEventDestroy(startTimer);
    hipEventDestroy(stopTimer);

    {
      // transfer matrix C back, from device to the host
      hipMemcpy(const_cast<float *>(C.data()), devC, C.size() * sizeof(float), hipMemcpyDeviceToHost);
    }

    {
      // clean gpu memory
      hipFree(devA);
      hipFree(devB);
      hipFree(devC);
    }
    return (config.kernelType == KernelType::KERNEL_CPU) ? cpuTime : gpuTime;
}
