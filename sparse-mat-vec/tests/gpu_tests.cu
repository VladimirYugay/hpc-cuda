#include "aux.h"
#include "kernels/kernels.h"
#include "test_aux.h"
#include "gmock/gmock.h"
#include "gtest/gtest.h"

using namespace ::testing;

class KernelsTests : public ::testing::Test {
  public:
    KernelsTests() : numRows(9) {

        y.resize(numRows, 0.0f);
        x.resize(numRows, 0.0f);
        for (int i = 0; i < x.size(); ++i) {
            x[i] = static_cast<float>(i + 1);
        }

        hipMalloc(&devX, numRows * sizeof(float));
        CHECK_ERR;
        hipMemcpy(devX, x.data(), numRows * sizeof(float), hipMemcpyHostToDevice);
        CHECK_ERR;

        hipMalloc(&devY, numRows * sizeof(float));
        CHECK_ERR;
    }

    ~KernelsTests() {
        hipFree(devX);
        CHECK_ERR;
        hipFree(devY);
        CHECK_ERR;
    }

  protected:
    void SetUp() override {}

    float *devX{nullptr};
    float *devY{nullptr};

    std::vector<float> x{};
    std::vector<float> y{};
    size_t numRows{};
};


//--------------------------------------------------------------------------------------------------
TEST_F(KernelsTests, CsrMvKernel) {
    CsrMatrix hostMatrix = get1DStencilCsrMatrix(numRows);
    DevCsrMatrix devMatrix;
    devMatrix.numRows = hostMatrix.numRows;
    devMatrix.nnz = hostMatrix.nnz;

    hipMalloc(&devMatrix.values, devMatrix.nnz * sizeof(float));
    CHECK_ERR;
    hipMemcpy(devMatrix.values, hostMatrix.values.data(), devMatrix.nnz * sizeof(float),
               hipMemcpyHostToDevice);
    CHECK_ERR;

    hipMalloc(&devMatrix.indices, devMatrix.nnz * sizeof(int));
    CHECK_ERR;
    hipMemcpy(devMatrix.indices, hostMatrix.indices.data(), devMatrix.nnz * sizeof(int),
               hipMemcpyHostToDevice);
    CHECK_ERR;

    hipMalloc(&devMatrix.start, (devMatrix.numRows + 1) * sizeof(int));
    CHECK_ERR;
    hipMemcpy(devMatrix.start, hostMatrix.start.data(), (devMatrix.numRows + 1) * sizeof(int),
               hipMemcpyHostToDevice);
    CHECK_ERR;


    launch_csrMatVecMult(devY, devMatrix, devX, ExecutionMode::PAGERANK);
    CHECK_ERR;

    std::vector<float> results(hostMatrix.numRows, 0.0f);
    hipMemcpy(const_cast<float *>(results.data()), devY, hostMatrix.numRows * sizeof(float),
               hipMemcpyDeviceToHost);
    CHECK_ERR;


    std::vector<float> expectedResult{0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, -10.0};
    EXPECT_THAT(results, ElementsAreArray(getExpectedArray(expectedResult)));

    hipFree(devMatrix.values);
    CHECK_ERR;
    hipFree(devMatrix.indices);
    CHECK_ERR;
    hipFree(devMatrix.start);
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
TEST_F(KernelsTests, CsrMvKernelVectorized) {
  CsrMatrix hostMatrix = get1DStencilCsrMatrix(numRows);
  DevCsrMatrix devMatrix;
  devMatrix.numRows = hostMatrix.numRows;
  devMatrix.nnz = hostMatrix.nnz;

  hipMalloc(&devMatrix.values, devMatrix.nnz * sizeof(float));
  CHECK_ERR;
  hipMemcpy(devMatrix.values, hostMatrix.values.data(), devMatrix.nnz * sizeof(float),
             hipMemcpyHostToDevice);
  CHECK_ERR;

  hipMalloc(&devMatrix.indices, devMatrix.nnz * sizeof(int));
  CHECK_ERR;
  hipMemcpy(devMatrix.indices, hostMatrix.indices.data(), devMatrix.nnz * sizeof(int),
             hipMemcpyHostToDevice);
  CHECK_ERR;

  hipMalloc(&devMatrix.start, (devMatrix.numRows + 1) * sizeof(int));
  CHECK_ERR;
  hipMemcpy(devMatrix.start, hostMatrix.start.data(), (devMatrix.numRows + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  CHECK_ERR;


  launch_csrMatVecMult(devY, devMatrix, devX, ExecutionMode::PAGERANK_VECTORIZED);
  CHECK_ERR;

  std::vector<float> results(hostMatrix.numRows, 0.0f);
  hipMemcpy(const_cast<float *>(results.data()), devY, hostMatrix.numRows * sizeof(float),
             hipMemcpyDeviceToHost);
  CHECK_ERR;


  std::vector<float> expectedResult{0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, -10.0};
  EXPECT_THAT(results, ElementsAreArray(getExpectedArray(expectedResult)));

  hipFree(devMatrix.values);
  CHECK_ERR;
  hipFree(devMatrix.indices);
  CHECK_ERR;
  hipFree(devMatrix.start);
  CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
TEST_F(KernelsTests, EllMvKernel) {
    EllMatrix hostMatrix;
    hostMatrix.numRows = numRows;
    hostMatrix.numColsPerRow = 4;

    // clang-format off
    std::vector<float> values{1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0,
                              0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 0.0,
                              0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 0.0,
                              0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 0.0};

    std::vector<int> indices {4, 6, 7, 1, 0, 6, 1, 5, 0,
                              0, 0, 0, 0, 1, 0, 0, 8, 0,
                              0, 0, 0, 0, 2, 0, 0, 0, 0,
                              0, 0, 0, 0, 3, 0, 0, 0, 0};
    // clang-format on

    DevEllMatrix devMatrix;
    devMatrix.numRows = hostMatrix.numRows;
    devMatrix.numColsPerRow = hostMatrix.numColsPerRow;

    const size_t memRequired = hostMatrix.numColsPerRow * hostMatrix.numRows;
    hipMalloc(&devMatrix.values, memRequired * sizeof(float));
    CHECK_ERR;
    hipMemcpy(devMatrix.values, values.data(), memRequired * sizeof(float),
               hipMemcpyHostToDevice);
    CHECK_ERR;

    hipMalloc(&devMatrix.indices, memRequired * sizeof(int));
    CHECK_ERR;
    hipMemcpy(devMatrix.indices, indices.data(), memRequired * sizeof(int),
               hipMemcpyHostToDevice);
    CHECK_ERR;

    launch_ellMatVecMult(devY, devMatrix, devX);
    CHECK_ERR;

    std::vector<float> results(hostMatrix.numRows, 0.0f);
    hipMemcpy(const_cast<float *>(results.data()), devY, hostMatrix.numRows * sizeof(float),
               hipMemcpyDeviceToHost);
    CHECK_ERR;


    std::vector<float> expectedResult{5.0, 7.0, 8.0, 2.0, 10.0, 7.0, 2.0, 15.0, 0.0};
    EXPECT_THAT(results, ElementsAreArray(getExpectedArray(expectedResult)));

    hipFree(devMatrix.values);
    CHECK_ERR;
    hipFree(devMatrix.indices);
    CHECK_ERR;
}


//--------------------------------------------------------------------------------------------------
TEST_F(KernelsTests, BandMvKernel) {
    EllMatrix hostMatrix;
    hostMatrix.numRows = numRows;
    hostMatrix.numColsPerRow = 3;

    // clang-format off
    std::vector<float> values{0.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0,
                              2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0,
                              -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, 0.0};
    // clang-format on

    DevBandMatrix devMatrix;
    devMatrix.numRows = hostMatrix.numRows;
    devMatrix.halfSize = 1;

    const size_t memRequired = hostMatrix.numColsPerRow * hostMatrix.numRows;
    hipMalloc(&devMatrix.values, memRequired * sizeof(float));
    CHECK_ERR;
    hipMemcpy(devMatrix.values, values.data(), memRequired * sizeof(float),
               hipMemcpyHostToDevice);
    CHECK_ERR;

    launch_bandMatVecMult(devY, devMatrix, devX);
    CHECK_ERR;

    std::vector<float> results(hostMatrix.numRows, 0.0f);
    hipMemcpy(const_cast<float *>(results.data()), devY, hostMatrix.numRows * sizeof(float),
               hipMemcpyDeviceToHost);
    CHECK_ERR;


    std::vector<float> expectedResult{0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 10.0};
    EXPECT_THAT(results, ElementsAreArray(getExpectedArray(expectedResult)));

    hipFree(devMatrix.values);
    CHECK_ERR;
}