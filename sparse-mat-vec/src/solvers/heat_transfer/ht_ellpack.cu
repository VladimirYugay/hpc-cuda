#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "ht.h"
#include "data_types.h"
#include "aux.h"
#include "kernels/kernels.h"


namespace ht {

void PoissonSolver::poisson_ellpack() {

    const size_t N = numGridPoints;
    // clang-format off
    float *devX{nullptr};
    hipMalloc(&devX, N * sizeof(float)); CHECK_ERR;
    hipMemcpy(devX, x.data(), N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;

    float *devY{nullptr};
    hipMalloc(&devY, N * sizeof(float)); CHECK_ERR;
    hipMemcpy(devY, y.data(), N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;

    EllMatrix matrix = get1DStencilEllMatrix(N);
    DevEllMatrix devMatrix;
    devMatrix.numColsPerRow = matrix.numColsPerRow;
    devMatrix.numRows = matrix.numRows;

    hipMalloc(&devMatrix.indices, N * matrix.numColsPerRow * sizeof(int)); CHECK_ERR;
    hipMemcpy(devMatrix.indices, matrix.indices.data(), N * matrix.numColsPerRow * sizeof(int),
               hipMemcpyHostToDevice); CHECK_ERR;


    hipMalloc(&devMatrix.values, N * matrix.numColsPerRow * sizeof(float)); CHECK_ERR;
    hipMemcpy(devMatrix.values, matrix.values.data(), N * matrix.numColsPerRow * sizeof(float),
               hipMemcpyHostToDevice); CHECK_ERR;
    // clang-format on


    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = nullptr;
    CUBLAS_CHECK(hipblasCreate(&cublasHandle));

    const float dx = 1.0f / (float)(N - 1);
    const float dt = (0.5f * dx * dx) / settings.conductivity;
    float err = std::numeric_limits<float>::max();
    CUBLAS_CHECK(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST));

    for (int counter = 0; ((err > settings.epsilon) && (settings.numIterations > counter)); ++counter) {
        launch_ellMatVecMult(devY, devMatrix, devX);

        // computes the Euclidean norm of the vector devY using cublas
        // TODO: T4.1 cuBLAS
        //CUBLAS_CHECK(cublas(...));

        float alpha = settings.conductivity * dt / (dx * dx);
      // TODO: T4.1: cuBLAS
        //CUBLAS_CHECK(cublas(...));

        if ((counter % settings.stepsToPrint) == 0) {
            {
                std::lock_guard<std::mutex> guard{plottingMutex};
                hipMemcpy(const_cast<float *>(x.data()), devX, N * sizeof(float),
                           hipMemcpyDeviceToHost);
                CHECK_ERR;
                status = PlottingStatus::UPDATED;
            }

            std::cout << "time = " << dt * static_cast<float>(counter) << ", err = " << err
                      << ", Temperature at x = 0.5: " << x[N / 2] << std::endl;
        }
    }

    {
        std::lock_guard<std::mutex> guard{plottingMutex};
        hipMemcpy(const_cast<float *>(x.data()), devX, N * sizeof(float),
                   hipMemcpyDeviceToHost);
        CHECK_ERR;
        status = PlottingStatus::UPDATED;
    }

    // clang-format off
    hipFree(devX); CHECK_ERR;
    hipFree(devY); CHECK_ERR;
    hipFree(devMatrix.indices); CHECK_ERR;
    hipFree(devMatrix.values); CHECK_ERR;
    CUBLAS_CHECK(hipblasDestroy(cublasHandle));
    // clang-format on
}
} // namespace ht