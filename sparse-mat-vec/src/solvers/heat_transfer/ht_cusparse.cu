#include "hip/hip_runtime.h"
#include "aux.h"
#include "ht.h"
#include "kernels/kernels.h"


namespace ht {
void PoissonSolver::poisson_cusparse() {

    const size_t N = numGridPoints;
    CsrMatrix matrix = get1DStencilCsrMatrix(N);

    DevCsrMatrix devMatrix;
    devMatrix.nnz = matrix.nnz;
    devMatrix.numRows = matrix.numRows;

    // clang-format off
    float *devX{nullptr};
    hipMalloc(&devX, N * sizeof(float)); CHECK_ERR;
    hipMemcpy(devX, x.data(), N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;

    float *devY{nullptr};
    hipMalloc(&devY, N * sizeof(float)); CHECK_ERR;
    hipMemcpy(devY, y.data(), N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;

    hipMalloc(&devMatrix.start, matrix.start.size() * sizeof(int)); CHECK_ERR;
    hipMemcpy(devMatrix.start, matrix.start.data(), matrix.start.size() * sizeof(int),
               hipMemcpyHostToDevice); CHECK_ERR;

    hipMalloc(&devMatrix.indices, matrix.indices.size() * sizeof(int)); CHECK_ERR;
    hipMemcpy(devMatrix.indices, matrix.indices.data(), matrix.indices.size() * sizeof(int),
               hipMemcpyHostToDevice); CHECK_ERR;

    hipMalloc(&devMatrix.values, matrix.values.size() * sizeof(float)); CHECK_ERR;
    hipMemcpy(devMatrix.values, matrix.values.data(), matrix.values.size() * sizeof(float),
               hipMemcpyHostToDevice); CHECK_ERR;
    // clang-format on


    hipblasHandle_t cublasHandle{nullptr};
    CUBLAS_CHECK(hipblasCreate(&cublasHandle));

    hipsparseHandle_t cusparseHandle{nullptr};
    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    hipsparseMatDescr_t descr{nullptr};
    CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
    CUSPARSE_CHECK(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CHECK(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    /*
    // Note: deprecated in cuda-sdk@11
    hipsparseHybMat_t hybridMatrix{nullptr};
    CUSPARSE_CHECK(hipsparseCreateHybMat(&hybridMatrix));
    CUSPARSE_CHECK(hipsparseScsr2hyb(cusparseHandle, devMatrix.numRows, devMatrix.numRows, descr,
                                    devMatrix.values, devMatrix.start, devMatrix.indices,
                                    hybridMatrix, 3, HIPSPARSE_HYB_PARTITION_MAX));
    */

    const float dx = 1.0f / (float)(N - 1);
    const float dt = (0.5f * dx * dx) / settings.conductivity;
    float err = std::numeric_limits<float>::max();
    for (int counter = 0; ((err > settings.epsilon) && (settings.numIterations > counter)); ++counter) {
        // launch_ellMatVecMult(devY, devMatrix, devX);
        float alpha = 1.0, beta = 0.0;
        // TODO: H4.1
        //CUSPARSE_CHECK(hipsparseShybmv(...));

        // computes the Euclidean norm of the vector devY using cublas
        // TODO: H4.1 similar to T4.1
        //CUBLAS_CHECK(cublas(...));

        alpha = dt / (dx * dx) * settings.conductivity;
        // TODO: H4.1 similar to T4.1
        //CUBLAS_CHECK(cublas(...));

        if ((counter % settings.stepsToPrint) == 0) {
            {
                std::lock_guard<std::mutex> guard{plottingMutex};
                hipMemcpy(const_cast<float *>(x.data()), devX, N * sizeof(float),
                           hipMemcpyDeviceToHost);
                CHECK_ERR;
                status = PlottingStatus::UPDATED;
            }

            std::cout << "time = " << dt * static_cast<float>(counter) << ", err = " << err
                      << ", Temperature at x = 0.5: " << x[N / 2] << std::endl;
        }
    }

    // clang-format off
    hipFree(devX); CHECK_ERR;
    hipFree(devY); CHECK_ERR;
    hipFree(devMatrix.start); CHECK_ERR;
    hipFree(devMatrix.indices); CHECK_ERR;
    hipFree(devMatrix.values); CHECK_ERR;

    // Note: deprecated in cuda-sdk@11
    // CUSPARSE_CHECK(hipsparseDestroyHybMat(hybridMatrix));
    CUSPARSE_CHECK(hipsparseDestroy(cusparseHandle));
    CUBLAS_CHECK(hipblasDestroy(cublasHandle));
    // clang-format on
}
} // namespace ht
